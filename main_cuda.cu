#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <algorithm>
#include <chrono>
#include <iterator>
#include <memory>
#include <random>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>

#include "util.hpp"

#define NDEBUG

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__constant__ int dSizes[1024]; // constant memory storing the dimensions

constexpr int numParticles = 32;

constexpr float cSwap = 0.6f; // probability

// deleter functor
template <typename T>
struct cuda_deleter {
  cuda_deleter() {}
  void operator()(T* ptr) {
    hipFree(ptr);
  }
};


__host__ __device__
int paddingTo32(int num) {
  return (num+31)/32*32;
}

template <typename T>
using cuda_unique_ptr = std::unique_ptr<T, decltype(cuda_deleter<T>())>;

// build graph in the memory according to the position relationship
__device__
void buildGraph(int& tIdx,
		int* HCG,
		int* VCG,
		int* OrderP, // __shared__
		int* OrderN, // __shared__
		int& rIncomeHCG, // register
		int& rIncomeVCG, // register
		const int numModules)
{
  rIncomeHCG = 0;
  rIncomeVCG = 0;
  for (auto j = 0; j < numModules; ++j) {
    int idx = numModules*paddingTo32(numModules)*(blockIdx.x) + j*paddingTo32(numModules) + tIdx; // edge j -> tIdx
    HCG[idx] = 0;
    VCG[idx] = 0;
    if (j != tIdx) {
      if (OrderN[j] < OrderN[tIdx]) {
	if (OrderP[j] < OrderP[tIdx]) {// left of
	  HCG[idx] = 1;
	  rIncomeHCG += 1;
	}
	else {// below
	  VCG[idx] = 1;
	  rIncomeVCG += 1;
	}
      }
    }
    __syncthreads();
  }
}

// topological visit graph, see report for detailed algorithm
// Note: Expect Qstart and Qend be 0
__device__
void topovisit(int& tIdx,
	       int& pIdx,
	       int* Graph,
	       int& Income, // count of income edge to vertex tIdx
	       bool& isAddedToQueue,
	       int* sCoord, // __shared__
	       int* Queue, // __shared__
	       int* path, // __shared__
	       int* pQstart, // ptr to __shared__
	       int* pQend, // ptr to __shared__
	       int* pDim, // ptr to __shared__
	       const int dOffset, // if width 0 height 512
	       const int numModules)
{
  //__syncthreads();
  // initialize Queue
  isAddedToQueue = false;
  sCoord[tIdx] = 0;
  Queue[tIdx] = -1;
  int idxQ = -1;
  __syncthreads();
  // if there is no incoming edge and it is not added to queue do that
  if ((Income == 0) && (!isAddedToQueue)) { 
    idxQ = atomicAdd(pQend, 1); // get an index in Q
    Queue[idxQ] = tIdx;
    isAddedToQueue = true;
  }
  __syncthreads();
  int Qstart = 0, Qend = 0;
  Qend = *pQend;
  __syncthreads();
  while (Qstart < Qend) { // there is element in Queue
    int currVertex = Queue[Qstart]; // a vertex with no incoming edge from edge not visited
#ifndef NDEBUG
    if (currVertex == -1) {
      atomicAdd(pQstart, 1);
      //printf("Q %d %d %d\n",pIdx, Qstart, Qend);
      //if (isAddedToQueue != false) {
      //printf("%d %d\n", tIdx, idxQ);
      //}
      //if (tIdx == 0) {
      //for (auto i = 0; i < *pQend; ++i) {
	  //printf("Q %d %d\n", i, Queue[i]);
      //}
	//}
      //__threadfence();
      //asm("trap;");
    }
    
    __syncthreads();

    if (*pQstart != 0) {
      printf("%d %d %d %d %d %d %d %d\n", pIdx, tIdx, currVertex, Qend, Qstart, Income, (int)isAddedToQueue, idxQ);
      __syncthreads();
      asm("trap;");
    }
#endif
    ++Qstart;
    __syncthreads();

    // set outgoing edge as visited by subtracting the count to income[tIdx]
    int idx = numModules*paddingTo32(numModules)*(blockIdx.x)
      + currVertex*paddingTo32(numModules) + tIdx; // edge currVertex -> tIdx
    if (Graph[idx] == 1) 
      Income -= 1;
    
    // get longest past from all predecents of currVertex
    path[tIdx] = 0;
    __syncthreads();
    idx = numModules*paddingTo32(numModules)*(blockIdx.x)
      + tIdx*paddingTo32(numModules) + currVertex; // edge tIdx -> currVertex
    if (Graph[idx] == 1) {
      path[tIdx] = sCoord[tIdx] + dSizes[tIdx+dOffset];
    }
    __syncthreads();

    // reduction to get max 
    for (auto i = numModules/2; i>0; i>>=1) {
      if ((tIdx <= i) && (tIdx+i < numModules)) 
	path[tIdx] = max(path[tIdx], path[tIdx+i]);
      __syncthreads();
    }
    // fix odd problem that last cycle is not performed
    if (tIdx == 0) 
      path[0] = max(path[0], path[1]);
      __syncthreads();
    // update Queue
    if ((Income == 0) && (!isAddedToQueue)) {
      int idxQ = atomicAdd(pQend, 1); // get an index in Q
      Queue[idxQ] = tIdx;
      isAddedToQueue = true;
    }
    __syncthreads();
    Qend = *pQend;
    __syncthreads();
    // increment, store result
    if (tIdx == 0) {
      *pQstart = 0;
      sCoord[currVertex] = path[0];
      *pDim = max(*pDim, path[0] + dSizes[currVertex+dOffset]);
    }
    __syncthreads();
  }
}

__global__
void initialization(int* lBestGammaP,
		    int* lBestGammaN,
		    int* GammaP,
		    int* GammaN,
		    int* lBestArea,
		    int* lastArea,
		    int* HCG,
		    int* VCG,
		    int* wC,
		    int* hC,
		    unsigned long long seed,
		    hiprandState* states,
		    const int numModules)
{
  int tIdx = threadIdx.x;
  int offset = blockIdx.x*paddingTo32(numModules);
  int pIdx = blockIdx.x;
  
  int chipArea=0;

  int rIncomeHCG=0, rIncomeVCG=0; // income count for this vertex
  bool isAddedToQueue=false;
  
  __shared__ int sGammaP[512];
  __shared__ int sGammaN[512];
  __shared__ int sOrderP[512];
  __shared__ int sOrderN[512];
  __shared__ int sWidCoord[512];
  __shared__ int sHeiCoord[512];
  __shared__ int queue[512];
  __shared__ int path[512];
  __shared__ int chipWidth, chipHeight;
  __shared__ int Qstart;
  __shared__ int Qend;

  hiprand_init(seed, tIdx+offset, 0, &states[tIdx+offset]);
  sOrderP[tIdx] = -1;
  sOrderN[tIdx] = -1;
  sGammaP[tIdx] = tIdx;
  sGammaN[tIdx] = tIdx;

  // random swap variables to initialize
  int stride = 1 << int(ceilf(log2f((float)numModules))-1);
  
  __syncthreads();
  for (; stride >=1; stride >>=1) { //
    if ((!((tIdx / stride) & 1)) && (tIdx+stride < numModules)) {// if it is a valid swap
      if (hiprand(&states[tIdx+offset]) & 1) {
	int temp = sGammaP[tIdx];
	sGammaP[tIdx] = sGammaP[tIdx+stride];
	sGammaP[tIdx+stride] = temp;
      }
      if (hiprand(&states[tIdx+stride]) & 1) {
	int temp = sGammaN[tIdx];
	sGammaN[tIdx] = sGammaN[tIdx+stride];
	sGammaN[tIdx+stride] = temp;
      }
    }
    __syncthreads();
  }
  
  sOrderP[sGammaP[tIdx]] = tIdx;
  sOrderN[sGammaN[tIdx]] = tIdx;
  __syncthreads();
  buildGraph(tIdx,
	     HCG,
	     VCG,
	     sOrderP,
	     sOrderN,
	     rIncomeHCG,
	     rIncomeVCG,
	     numModules);
  __syncthreads();

  if (tIdx == 0) {
    Qstart = 0;
    Qend = 0;
    chipWidth = 0;
    chipHeight = 0;
  }
  __syncthreads();
  topovisit(tIdx,
	    pIdx,
	    HCG,
	    rIncomeHCG,
	    isAddedToQueue,
	    sWidCoord, // __shared__
	    queue, // __shared__
	    path, // __shared__
	    &Qstart, // ptr to __shared__
	    &Qend, // ptr to __shared__
	    &chipWidth, // ptr to __shared__
	    0, // if width 0 height 512
	    numModules);

  if (tIdx == 0) {
    if (Qend != numModules)
      printf("Unexpected end at %d\n", Qend);
    Qstart = 0;
    Qend = 0;
  }
  __syncthreads();
  
  topovisit(tIdx,
	    pIdx,
	    VCG,
	    rIncomeVCG,
	    isAddedToQueue,
	    sHeiCoord, // __shared__
	    queue, // __shared__
	    path, // __shared__
	    &Qstart, // ptr to __shared__
	    &Qend, // ptr to __shared__
	    &chipHeight, // ptr to __shared__
	    512, // if width 0 height 512
	    numModules);
  
  if (tIdx == 0) {
    if (Qend != numModules)
      printf("Unexpected end at %d\n", Qend);
  }
  
  __syncthreads();
  wC[tIdx+offset] = sWidCoord[tIdx];
  hC[tIdx+offset] = sHeiCoord[tIdx];
  lBestGammaP[tIdx+offset] = sGammaP[tIdx];
  lBestGammaN[tIdx+offset] = sGammaN[tIdx];
  GammaP[tIdx+offset] = sGammaP[tIdx];
  GammaN[tIdx+offset] = sGammaN[tIdx];
  if (tIdx == 0) {
    chipArea = chipWidth*chipHeight;
    lastArea[pIdx] = chipArea;
    lBestArea[pIdx] = chipArea;
  }
}

// function used to perform random swap to gamma sequence
__device__
void swapSequence(int& tIdx,
		  int sourceOffset,
		  int targetOffset, // globalbest ones should be 0 here
		  int* swapTo, // __share__ var
		  int* sSource, // __shared__ array
		  int* sTarget, // __shared__ array
		  int* sGamma, // __shared, this is the swap taking place
		  int* Gamma, // global, only read
		  int* TargetGamma, // global
		  hiprandState* states,
		  const int numModules)
{
  sSource[tIdx] = Gamma[tIdx+sourceOffset];
  sTarget[tIdx] = TargetGamma[tIdx+targetOffset];
  __syncthreads();
  for (auto i=0; i<numModules; ++i) {
    if (sTarget[i] == sSource[tIdx])
      *swapTo = tIdx;
    __syncthreads();
    if (tIdx == 0) // let thread 0 swap
      if (i != tIdx) { // only swap if the id is different
	int temp = sSource[*swapTo];
	sSource[*swapTo] = sSource[i];
	sSource[i] = temp;
	if (hiprand_uniform(&states[sourceOffset]) < cSwap) {
	  temp = sGamma[*swapTo];
	  sGamma[*swapTo] = sGamma[i];
	  sGamma[i] = temp;
	}
      }
    __syncthreads();
  }
}

__global__
void update(int* GammaP,
	    int* GammaN,
	    int* lBestGammaP,
	    int* lBestGammaN,
	    int* gBestGammaP,
	    int* gBestGammaN,
	    int* HCG,
	    int* VCG,
	    int* wC,
	    int* hC,
	    int* lBestArea,
	    int* lastArea,
	    hiprandState* states,
	    const int numModules)
{
  int tIdx = threadIdx.x;
  int offset = blockIdx.x*paddingTo32(numModules);
  int pIdx = blockIdx.x;
  
  int chipArea=0;

  int rIncomeHCG=0, rIncomeVCG=0; // income count for this vertex
  bool isAddedToQueue=false;
  int cSwapRandom = ceilf(0.01*numModules);
  
  __shared__ int sGammaP[512];
  __shared__ int sGammaN[512];
  __shared__ int sSource[512];
  __shared__ int sTarget[512];
  __shared__ int sOrderP[512];
  __shared__ int sOrderN[512];
  __shared__ int sWidCoord[512];
  __shared__ int sHeiCoord[512];
  __shared__ int queue[512];
  __shared__ int path[512];
  __shared__ int chipWidth, chipHeight;
  __shared__ int swapTo;
  __shared__ int Qstart;
  __shared__ int Qend;

  // update velocity and search space

  sGammaP[tIdx] = GammaP[tIdx+offset];
  sGammaN[tIdx] = GammaN[tIdx+offset];
  
  // local Positive swap
  swapSequence(tIdx,
	       offset,
	       offset,
	       &swapTo,
	       sSource, // __shared__
	       sTarget, // __shared__
	       sGammaP, // __shared, this is the swap taking place
	       GammaP, // global mem, only read
	       lBestGammaP, // global mem
	       states,
	       numModules);
  __syncthreads();

  // local Negative swap
  swapSequence(tIdx,
	       offset,
	       offset,
	       &swapTo,
	       sSource, // __shared__
	       sTarget, // __shared__
	       sGammaN, // __shared, this is the swap taking place
	       GammaN, // global mem, only read
	       lBestGammaN, // global mem
	       states,
	       numModules);
  __syncthreads();

  // global Positive swap
  swapSequence(tIdx,
	       offset,
	       0,
	       &swapTo,
	       sSource, // __shared__
	       sTarget, // __shared__
	       sGammaP, // __shared, this is the swap taking place
	       GammaP, // global mem, only read
	       gBestGammaP, // global mem
	       states,
	       numModules);
  __syncthreads();

  // global Negative swap
  swapSequence(tIdx,
	       offset,
	       0,
	       &swapTo,
	       sSource, // __shared__
	       sTarget, // __shared__
	       sGammaN, // __shared, this is the swap taking place
	       GammaN, // global mem, only read
	       gBestGammaN, // global mem
	       states,
	       numModules);
  __syncthreads();

  // add some random swap
  if (tIdx == 0) 
    for (auto i=0; i < cSwapRandom; ++i) {
      unsigned int j = hiprand(&states[tIdx+offset]) % numModules;
      unsigned int k = hiprand(&states[tIdx+offset]) % numModules;
      unsigned int type = hiprand(&states[tIdx+offset]) % 3;
      int temp = 0;
      if (type != 1) {
	temp = sGammaP[j];
	sGammaP[j] = sGammaP[k];
	sGammaP[k] = temp;
      }
      if (type != 2) {
	temp = sGammaN[j];
	sGammaN[j] = sGammaN[k];
	sGammaN[k] = temp;
      }
    }

  
  // build Sequence Pair

  __syncthreads();
  sOrderP[tIdx] = -1;
  sOrderN[tIdx] = -1;
  __syncthreads();
  GammaP[tIdx+offset] = sGammaP[tIdx];
  GammaN[tIdx+offset] = sGammaN[tIdx];
  sOrderP[sGammaP[tIdx]] = tIdx;
  sOrderN[sGammaN[tIdx]] = tIdx;
  __syncthreads();
  if ((sOrderP[tIdx] == -1) || (sOrderN[tIdx] == -1)) {
    __threadfence();
    asm("trap;");
  }
  
  __syncthreads();

  buildGraph(tIdx,
	     HCG,
	     VCG,
	     sOrderP,
	     sOrderN,
	     rIncomeHCG,
	     rIncomeVCG,
	     numModules);
  __syncthreads();

  if (tIdx == 0) {
    Qstart = 0;
    Qend = 0;
    chipWidth = 0;
    chipHeight = 0;
  }
  __syncthreads();
  topovisit(tIdx,
	    pIdx,
	    HCG,
	    rIncomeHCG,
	    isAddedToQueue,
	    sWidCoord, // __shared__
	    queue, // __shared__
	    path, // __shared__
	    &Qstart, // ptr to __shared__
	    &Qend, // ptr to __shared__
	    &chipWidth, // ptr to __shared__
	    0, // if width 0 height 512
	    numModules);

  if (tIdx == 0) {
    
    if (Qend != numModules) {
      printf("1Unexpected end at %d, %d\n", Qend, Qstart);
    }
    Qstart = 0;
    Qend = 0;
  }
  __syncthreads();
  
  topovisit(tIdx,
	    pIdx,
	    VCG,
	    rIncomeVCG,
	    isAddedToQueue,
	    sHeiCoord, // __shared__
	    queue, // __shared__
	    path, // __shared__
	    &Qstart, // ptr to __shared__
	    &Qend, // ptr to __shared__
	    &chipHeight, // ptr to __shared__
	    512, // if width 0 height 512
	    numModules);
  
  if (tIdx == 0) {
    if (Qend != numModules) {
      printf("2Unexpected end at %d, %d\n", Qend, Qstart);
    }
  }
  
  __syncthreads();
  chipArea = chipWidth*chipHeight;
  wC[tIdx+offset] = sWidCoord[tIdx];
  hC[tIdx+offset] = sHeiCoord[tIdx];
  
  // update local best
  if (chipArea < lBestArea[pIdx]) {
    if (tIdx == 0) 
      lBestArea[pIdx] = chipArea;
    lBestGammaP[tIdx+offset] = sGammaP[tIdx];
    lBestGammaN[tIdx+offset] = sGammaN[tIdx];
  }
  if (tIdx == 0) 
    lastArea[pIdx] = chipArea;
}

__global__
void copyGlobalBest(int pIdx,
		    int* gBestGammaP,
		    int* gBestGammaN,
		    int* GammaP,
		    int* GammaN,
		    int* gBestWidC,
		    int* gBestHeiC,
		    int* wC,
		    int* hC,
		    const int numModules)
{
  int tIdx = threadIdx.x;
  int offset = pIdx*paddingTo32(numModules);
  gBestGammaP[tIdx] = GammaP[tIdx+offset];
  gBestGammaN[tIdx] = GammaN[tIdx+offset];
  gBestWidC[tIdx] = wC[tIdx+offset];
  gBestHeiC[tIdx] = hC[tIdx+offset];
}

int main(int argc, const char *argv[])
{
  // Timing
  using Time = std::chrono::high_resolution_clock;
  using us = std::chrono::microseconds;
  using fsec = std::chrono::duration<float>;


  // parameter parsing
  std::vector<std::string> args(argv, argv+argc);
  int numModules;
  std::vector<int> widths, heights;
  std::vector<int> sizes(1024, 0);
  try {
    std::ifstream ckt_file(args.at(1));
    if (!ckt_file.is_open()) {
      std::cerr << "Cannot open file:\t" << args.at(1) << std::endl;
      exit(1);
    }
    read_ckt(ckt_file, numModules, widths, heights);
    ckt_file.close();
    std::copy(std::begin(widths), std::end(widths), std::begin(sizes));
    std::copy(std::begin(heights), std::end(heights), std::begin(sizes)+512);
  } catch (const std::out_of_range& e) {
    std::cerr << "Not enough parameters." << std::endl;
    printUsage(args.at(0));
    exit(1);
  }
  // Allocate memory needed
  auto myCudaMalloc = [](size_t size)
    {
     void* ptr;
     hipMalloc(&ptr, size);
     return ptr;
    };
  auto myCudaMallocManaged = [](size_t size)
    {
     void* ptr;
     hipMallocManaged(&ptr, size);
     return ptr;
    };
  hipSetDevice(0);
  hipFree(0);
  std::ofstream result_file("result.txt");
  auto start = Time::now();
  // global scope
  cuda_unique_ptr<int> gBestGammaP((int*)myCudaMallocManaged(numModules*sizeof(int)));
  cuda_unique_ptr<int> gBestGammaN((int*)myCudaMallocManaged(numModules*sizeof(int)));
  cuda_unique_ptr<int> gBestWidC((int*)myCudaMallocManaged(numModules*sizeof(int)));
  cuda_unique_ptr<int> gBestHeiC((int*)myCudaMallocManaged(numModules*sizeof(int)));
  int gBestArea = -1;
  // padded memories, local to block
  int padded = paddingTo32(numModules);
  cuda_unique_ptr<int> lBestGammaP((int*)myCudaMalloc(padded*numParticles*sizeof(int)));
  cuda_unique_ptr<int> lBestGammaN((int*)myCudaMalloc(padded*numParticles*sizeof(int)));
  cuda_unique_ptr<int> GammaP((int*)myCudaMalloc(padded*numParticles*sizeof(int)));
  cuda_unique_ptr<int> GammaN((int*)myCudaMalloc(padded*numParticles*sizeof(int)));
  cuda_unique_ptr<int> wC((int*)myCudaMalloc(padded*numParticles*sizeof(int)));
  cuda_unique_ptr<int> hC((int*)myCudaMalloc(padded*numParticles*sizeof(int)));
  cuda_unique_ptr<hiprandState> states((hiprandState*)myCudaMalloc(padded*numParticles*sizeof(hiprandState)));
  // storage for HCG and VCG
  cuda_unique_ptr<int> HCG((int*)myCudaMalloc(numModules*padded*numParticles*sizeof(int)));
  cuda_unique_ptr<int> VCG((int*)myCudaMalloc(numModules*padded*numParticles*sizeof(int)));
  // local to block, each block maintain one
  cuda_unique_ptr<int> lastArea((int*)myCudaMallocManaged(numParticles*sizeof(int)));
  cuda_unique_ptr<int> lBestArea((int*)myCudaMalloc(numParticles*sizeof(int)));


  hipMemcpyToSymbol(HIP_SYMBOL(dSizes), sizes.data(), 1024*sizeof(int));
  
  
  
  // initialize velocity and search space with random variable
  std::random_device rd;

  initialization<<<numParticles, numModules>>>(lBestGammaP.get(),
					       lBestGammaN.get(),
					       GammaP.get(),
					       GammaN.get(),
					       lBestArea.get(),
					       lastArea.get(),
					       HCG.get(),
					       VCG.get(),
					       wC.get(),
					       hC.get(),
					       rd(),
					       states.get(),
					       numModules);
  
  hipDeviceSynchronize();
  gpuErrchk( hipPeekAtLastError() );
  int minParIdx = std::distance(lastArea.get(),
				std::min_element(lastArea.get(), lastArea.get()+numParticles));

  gBestArea = lastArea.get()[minParIdx];
  copyGlobalBest<<<1, numModules>>>(minParIdx,
				    gBestGammaP.get(),
				    gBestGammaN.get(),
				    GammaP.get(),
				    GammaN.get(),
				    gBestWidC.get(),
				    gBestHeiC.get(),
				    wC.get(),
				    hC.get(),
				    numModules);   
  //for (auto i = 0; i < numParticles; ++i) 
  //std::cout << lastArea.get()[i] << "\t";
  std::cout << std::endl;
  hipDeviceSynchronize();
  std::cout << gBestArea << std::endl;

  //  std::cout << "Best at this round appears at pIdx:" << minParIdx
  //	    << "\tWith Area:" << gBestArea << std::endl;
  int counter = 0;
  int cycle = 0;
  while (counter < 100) {
    update<<<numParticles, numModules>>>(GammaP.get(),
					 GammaN.get(),
					 lBestGammaP.get(),
					 lBestGammaN.get(),
					 gBestGammaP.get(),
					 gBestGammaN.get(),
					 HCG.get(),
					 VCG.get(),
					 wC.get(),
					 hC.get(),
					 lBestArea.get(),
					 lastArea.get(),
					 states.get(),
					 numModules);
    hipDeviceSynchronize();
    gpuErrchk( hipPeekAtLastError() );
    minParIdx = std::distance(lastArea.get(),
			      std::min_element(lastArea.get(), lastArea.get()+numParticles));    
    ++counter;
    if (gBestArea > lastArea.get()[minParIdx]) {
      counter = 0;
      gBestArea = lastArea.get()[minParIdx];
      copyGlobalBest<<<1, numModules>>>(minParIdx,
					gBestGammaP.get(),
					gBestGammaN.get(),
					GammaP.get(),
					GammaN.get(),
					gBestWidC.get(),
					gBestHeiC.get(),
					wC.get(),
					hC.get(),
					numModules);
      std::cout << cycle << "\t" << gBestArea << std::endl; 
      hipDeviceSynchronize();
    }
    //for (auto i = 0; i < numParticles; ++i) 
    // std::cout << lastArea.get()[i] << "\t";
    //std::cout << std::endl;  
    
    ++cycle;
  }
  std::cout << "Best Area:" << gBestArea  << "\tUsing "<< cycle << " Cycles"<< std::endl;
  //for (auto i = 0; i < numModules; ++i) 
  //std::cout << i << "\t" << gBestGammaP.get()[i] << "\t" << gBestGammaN.get()[i] << std::endl;
  // Timing
  auto stop = Time::now();
  fsec fs = stop - start;
  us d = std::chrono::duration_cast<us>(fs);
  std::cout << "Program took \t" << fs.count() << "s" << std::endl
	    << "\tor \t" << d.count() << "us" << std::endl;
  // Note: TURN OFF UNIFIED MEMORY PROFILE
  std::vector<int>
    gBestGammaP_h(numModules),
    gBestGammaN_h(numModules),
    gBestWidC_h(numModules),
    gBestHeiC_h(numModules);
  std::copy(gBestGammaP.get(), gBestGammaP.get()+numModules, std::begin(gBestGammaP_h));
  std::copy(gBestGammaN.get(), gBestGammaN.get()+numModules, std::begin(gBestGammaN_h));
  std::copy(gBestWidC.get(), gBestWidC.get()+numModules, std::begin(gBestWidC_h));
  std::copy(gBestHeiC.get(), gBestHeiC.get()+numModules, std::begin(gBestHeiC_h));
  write_ckt(result_file, gBestArea, numModules, gBestGammaP_h, gBestGammaN_h, gBestWidC_h, gBestHeiC_h);
  return 0;
}
